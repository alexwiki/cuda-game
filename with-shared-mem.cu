#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define M 32
#define KNRM  "\x1B[0m"
#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KYEL  "\x1B[33m"
#define KBLU  "\x1B[34m"
#define KMAG  "\x1B[35m"
#define KCYN  "\x1B[36m"
#define KWHT  "\x1B[37m"


__global__ void uni_func(int *A,int width,int *OUT)
{
	__shared__ int ns[32*32];//neighboors state
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	bool first_row,last_row,first_col,last_col;
	first_row = col>0 && col < width-1;//prwti grammi = 0
	last_row = (col>(width*width)-width) && (col<(width*width)-1) ;//teleutaia grammi = 0
	first_col = col%width == 0;//prwti stili = 0
	last_col = col%width == width -1 ;
	if (!(first_row || last_row)){// den einai "orizontio" kadro
		//kai ta stoixeia sto last_row - last_col prepei na feroun ton panw kai katw geitona tous
		ns[col] = A[col];
		ns[col-width] = A[col-width];
		ns[col+width] = A[col+width];
		__syncthreads();//kathe stoixeio fernei 3 anthrwpous(2 neigh and me) kai arazei....

		if (!(last_col || first_col)){//den einai "katheto" kadro
		//mono ta stoixeia pou den einai se kadro prepei na elenksoun geitones...

						int n[8];
						n[0] = ns[(col-1-width)] ;
						n[1] = ns[(col-width)] ;
						n[2] = ns[(col+1-width)] ;

					//	row=2;
						//kentriki grammi tou 3x3
						n[3] = ns[(col-1)] ;//ditikos geitonas
					//	= ns[row*width + col] ; // κεντρικο κελι
						int iam = ns[col] ; // κεντρικο κελι
						n[4] = ns[(col+1)] ;//anatolikos geitonas

					//	row =3 ;
						n[5] = ns[(col-1+width)] ;
						n[6] = ns[(col+width)] ;
						n[7] = ns[(col+1+width)] ;


							//on || off || dying
							//Ξεκιναμε να οριζουμε τις συνθηκες αλλαγης καταστασεων:
							int counter_alive=0;
							int counter_dead=0;		// οι 3 μετρητες μας που θα πρεπει να γυρισουν
							int counter_DYING=0;	//στην CPU και θα εκτυπωθουν

							// rules: -1: dying && 0:off && 1:on
						//Στον παρακατω κωδικα μετραμε του alive ,dead ,DYING
						// tsekaroume ean edw einai to lathos ston kwdika

							for (int i = 0; i <= 7; i++)
							{
								if (n[i] != -1)//for sure is not dying - actually is not -1(negative number)
								{
									counter_alive += n[i];//counter_alive = counter_alive + 0/1
								}
								else//
								{
									counter_DYING -= n[i] ;//-0 || -(-1)=+1
								}
							}
							counter_dead = 8 - ( counter_alive + counter_DYING);//all neighboors - not_dying



							if(iam == -1)//i am dying
							{
								iam = 0;//i am off
							}
							else if(iam == 1)//i am on
							{
								iam = -1;	//i am dying
							}
							else if(iam == 0 && counter_alive == 2 )//i am off and 2 neighboors on
							{
								iam = 1;	//i will be on


							}


							OUT[col] = iam;

			}
			else{//einai to katheto kadro
					OUT[ col] = 0;
			}
		
	}
	else{//einai to orizontio kadro
		OUT[ col] = 0;
	}

}

int main() {
	//initialize A
	int i,j;
	int on=0;
	int off=0;
	int dying=0;
	//int  M =  32;
	int N=M*M;//all elements of A
	int A[M][M] ;
	int OUT[M][M] ;
	srand (time(NULL));
	printf("\n....IN MAIN...\n");
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (i==0 || i==M-1 || j==M-1 || j==0){
				A[i][j] = 0;//to perigramma tou pinaka
				OUT[i][j] = 0;
			}
			else{
				A[i][j]=  rand()%3 -1;
				//if (A[i][j] == -1){printf("%d   ", A[i][j]);}
				//else{printf(" %d   ", A[i][j]);}
				OUT[i][j] = -9;
			}
		}
		//printf("\n");
	}
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (A[i][j] == -1){printf("%d ", A[i][j]);}
			else{printf(" %d ", A[i][j]);}
		}
		printf("\n");
	}
	//launching kernel

	int *A_device;
	//int A_size = N*sizeof(int) ;
	const size_t A_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&A_device, A_size);

	int *OUT_device;
	//int A_size = N*sizeof(int) ;
	const size_t OUT_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&OUT_device, OUT_size);

	hipMemcpy(A_device, A, A_size, hipMemcpyHostToDevice);
	hipMemcpy(OUT_device, OUT, OUT_size, hipMemcpyHostToDevice);


	//the game is on Mrs. Hatson :)

	int turn = 0;

	while (1){

		if (turn % 2 == 0){//zigos arithmos seiras: A->in, Out->Out
			uni_func<<<M,M>>>(A_device,M,OUT_device);
			hipMemcpy(OUT, OUT_device, A_size,  hipMemcpyDeviceToHost);//thats work
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (OUT[i][j] == -1){printf("%s%d ",KRED, OUT[i][j]);}
					else if (OUT[i][j] == 1){printf(" %s%d ",KGRN, OUT[i][j]);}
					else{printf(" %s%d ",KNRM, OUT[i][j]);}

					//make counter
					if (OUT[i][j] == -1){ dying++;}
					else if (OUT[i][j] == 1) {on++;}
					else {off++;}


				}
				printf("\n");
			}
		}
		else{
			uni_func<<<M,M>>>(OUT_device,M,A_device);
			hipMemcpy(A, A_device, A_size,  hipMemcpyDeviceToHost);
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (A[i][j] == -1){printf("%s%d ",KRED, A[i][j]);}
					else if (A[i][j]==1){printf(" %s%d ",KGRN, A[i][j]);}
					else {printf(" %s%d ",KNRM, A[i][j]);}

					//make counter
					if (A[i][j] == -1){ dying++;}
					else if (A[i][j] == 1) {on++;}
					else {off++;}
				}
				printf("\n");
			}
		}
		//print counter
		printf("\n%s----------------------------------------------------\n",KNRM);
		printf("counter_alive: %d, counter_dying: %d, counter_dead: %d\n",on,dying,off);
		printf("--------------------------------------------------------\n");
		//counters = 0
		if (off == N){break;}//all elements are off (N=M*M)
		on = 0;
		off = 0;
		dying = 0;
		turn++;//auksanoume seira gia na kalesoume uni_func me allagi eisodwn-eksodwn


	}



	return 0;
}
