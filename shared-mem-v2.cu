#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define M 32
#define KNRM  "\x1B[0m"
#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KYEL  "\x1B[33m"
#define KBLU  "\x1B[34m"
#define KMAG  "\x1B[35m"
#define KCYN  "\x1B[36m"
#define KWHT  "\x1B[37m"


__global__ void uni_func(int *A,int width,int *OUT)
{
	__shared__ int ns[32*32];//neighboors state
	int col = blockIdx.x*blockDim.x + threadIdx.x;
//	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int tid =  col;
	//unsigned int tid = threadIdx.x;
	//uni_1d[row*width + col] = me
/*
	ns[tid] = A[row*width + col];
	__syncthreads();


//	int iam = A[row*width + col] ; // κεντρικο κελι

	int iam = ns[tid];
	int n[8];//neighboors

	n[0] = ns[tid-M-1] ;
	n[1] = ns[tid-M] ;
	n[2] = ns[tid-M+1] ;

	n[3] = ns[tid-1] ;
	n[4] = ns[tid+1] ;

	n[5] = ns[tid+M-1] ;
	n[6] = ns[tid+M] ;
	n[7] = ns[tid+M+1] ;
	*/

	bool first_row,last_row,first_col,last_col;
	first_row = col>=0 && col <= width-1;//prwti grammi = 0
	last_row = (col>=(width*width)-width) && (col<=(width*width)-1) ;//teleutaia grammi = 0
	first_col = col%width == 0;//prwti stili = 0
	last_col = col%width == width -1 ;
	if (!(first_row || last_row)){// || first_col || last_col )) {//oxi kadro tou pinaka

			if (tid%2==0){//mono ziga thread fernoun kosmo
				ns[row*width + col] = A[row*width + col];//me
				ns[(row-1)*width + (col)] = A[(row-1)*width + (col)];
				ns[(row-1)*width + (col+1)] = A[(row-1)*width + (col+1)];
				ns[row*width + (col+1)] = A[row*width + (col+1)];
				ns[(row+1)*width + (col)] = A[(row+1)*width + (col)];
				ns[(row+1)*width + (col+1)] = A[(row+1)*width + (col+1)];
			}

			__syncthreads();

			int iam = ns[row*width + col];
			int n[8];//neighboors

			n[0] = ns[(row-1)*width + (col-1)] ;
			n[1] = ns[(row-1)*width + (col)] ;
			n[2] = ns[(row-1)*width + (col+1)] ;

			n[3] = ns[row*width + (col-1)] ;
			n[4] = ns[row*width + (col+1)] ;

			n[5] = ns[(row+1)*width + (col-1)] ;
			n[6] = ns[(row+1)*width + (col)] ;
			n[7] = ns[(row+1)*width + (col+1)] ;


			//on || off || dying
			//Ξεκιναμε να οριζουμε τις συνθηκες αλλαγης καταστασεων:
			int counter_alive=0;
			int counter_dead=0;		// οι 3 μετρητες μας που θα πρεπει να γυρισουν
			int counter_DYING=0;	//στην CPU και θα εκτυπωθουν

			// rules: -1: dying && 0:off && 1:on
		//Στον παρακατω κωδικα μετραμε του alive ,dead ,DYING
		// tsekaroume ean edw einai to lathos ston kwdika

			for (int i = 0; i <= 7; i++)
			{
				if (n[i] != -1)//for sure is not dying - actually is not -1(negative number)
				{
					counter_alive += n[i];//counter_alive = counter_alive + 0/1
				}
				else//
				{
					counter_DYING -= n[i] ;//-0 || -(-1)=+1
				}
			}
			counter_dead = 8 - ( counter_alive + counter_DYING);//all neighboors - not_dying



			if(iam == -1)//i am dying
			{
				iam = 0;//i am off
			}
			else if(iam == 1)//i am on
			{
			 	iam = -1;	//i am dying
			}
			else if(iam == 0 && counter_alive == 2 )//i am off and 2 neighboors on
			{
				iam = 1;	//i will be on


			}

		  if (first_col || last_col ){//sto kadro tou pinaka
				iam = 0;
			}

			OUT[row*width + col] = iam;




			//twra pou to skeutomai to perigramma tou pinaka = 0 mporei na metaferthei kai
			//sto host wste na min gemizoume me if ta thread .... ;)
			//sizita to me alex

	}
	else{
		OUT[row*width + col] = 0;
	}

}

int main() {
	//initialize A
	int i,j;
	int on=0;
	int off=0;
	int dying=0;
	//int  M =  32;
	int N=M*M;//all elements of A
	int A[M][M] ;
	int OUT[M][M] ;
	srand (time(NULL));
	printf("\n....IN MAIN...\n");
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (i==0 || i==M-1 || j==M-1 || j==0){
				A[i][j] = 0;//to perigramma tou pinaka
				OUT[i][j] = 0;
			}
			else{
				A[i][j]=  rand()%3 -1;
				//if (A[i][j] == -1){printf("%d   ", A[i][j]);}
				//else{printf(" %d   ", A[i][j]);}
				OUT[i][j] = -9;
			}
		}
		//printf("\n");
	}
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (A[i][j] == -1){printf("%d ", A[i][j]);}
			else{printf(" %d ", A[i][j]);}
		}
		printf("\n");
	}
	//launching kernel

	int *A_device;
	//int A_size = N*sizeof(int) ;
	const size_t A_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&A_device, A_size);

	int *OUT_device;
	//int A_size = N*sizeof(int) ;
	const size_t OUT_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&OUT_device, OUT_size);

	hipMemcpy(A_device, A, A_size, hipMemcpyHostToDevice);
	hipMemcpy(OUT_device, OUT, OUT_size, hipMemcpyHostToDevice);


	//the game is on Mrs. Hatson :)

	int turn = 0;

	while (1){

		if (turn % 2 == 0){//zigos arithmos seiras: A->in, Out->Out
			uni_func<<<M,M>>>(A_device,M,OUT_device);
			hipMemcpy(OUT, OUT_device, A_size,  hipMemcpyDeviceToHost);//thats work
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (OUT[i][j] == -1){printf("%s%d ",KRED, OUT[i][j]);}
					else if (OUT[i][j] == 1){printf(" %s%d ",KGRN, OUT[i][j]);}
					else{printf(" %s%d ",KNRM, OUT[i][j]);}

					//make counter
					if (OUT[i][j] == -1){ dying++;}
					else if (OUT[i][j] == 1) {on++;}
					else {off++;}


				}
				printf("\n");
			}
		}
		else{
			uni_func<<<M,M>>>(OUT_device,M,A_device);
			hipMemcpy(A, A_device, A_size,  hipMemcpyDeviceToHost);
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (A[i][j] == -1){printf("%s%d ",KRED, A[i][j]);}
					else if (A[i][j]==1){printf(" %s%d ",KGRN, A[i][j]);}
					else {printf(" %s%d ",KNRM, A[i][j]);}

					//make counter
					if (A[i][j] == -1){ dying++;}
					else if (A[i][j] == 1) {on++;}
					else {off++;}
				}
				printf("\n");
			}
		}
		//print counter
		printf("\n%s----------------------------------------------------\n",KNRM);
		printf("counter_alive: %d, counter_dying: %d, counter_dead: %d\n",on,dying,off);
		printf("--------------------------------------------------------\n");
		//counters = 0
		if (off == N){break;}//all elements are off (N=M*M)
		on = 0;
		off = 0;
		dying = 0;
		turn++;//auksanoume seira gia na kalesoume uni_func me allagi eisodwn-eksodwn


	}



	return 0;
}
